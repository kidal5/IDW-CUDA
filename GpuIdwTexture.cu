#include "hip/hip_runtime.h"
#include "GpuIdwTexture.cuh"

#include "hip/hip_runtime_api.h"
#include "surface_functions.h"
#include "surface_indirect_functions.h"

#include <cuda_gl_interop.h>

static void handleCudaError(const hipError_t error, const char* file, const int line) {
	if (error == hipSuccess) return;

	fmt::print("{} in {} at line {}\n", hipGetErrorString(error), file, line);
	exit(EXIT_FAILURE);
}

#define CHECK_ERROR( error ) ( handleCudaError( error, __FILE__, __LINE__ ) )

namespace
{
	__global__ void gpuDrawAnchorPointsKernel(hipSurfaceObject_t surfObject, const int* anchorPoints, const int anchorPointsCount, const int width, const int height) {

		const int x = threadIdx.x;

		if (x < anchorPointsCount) {
			const int xAnchor = anchorPoints[3 * x];
			const int yAnchor = anchorPoints[3 * x + 1];

			uchar1 data;
			surf2Dread(&data, surfObject, xAnchor + 1, yAnchor);
			data.x = data.x > 127 ? 0 : 255;

			for (int shiftX = -1; shiftX < 1; shiftX++) {
				for (int shiftY = -1; shiftY < 1; shiftY++) {
					surf2Dwrite(data, surfObject, xAnchor + shiftX , yAnchor + shiftY);
				}
			}
		}
	}
	
	__device__ double computeWiGpu(const int ax, const int ay, const int bx, const int by, const double pParam) {
		const float dist = sqrtf((ax - bx) * (ax - bx) + (ay - by) * (ay - by));
		return 1 / powf(dist, pParam);
	}

	__global__ void gpuTextureKernel(hipSurfaceObject_t surfObject, const int* anchorPoints, const int anchorPointsCount, const double pParam, const int width, const int height)
	{
		const int x = blockIdx.x * blockDim.x + threadIdx.x;
		const int y = blockIdx.y * blockDim.y + threadIdx.y;
		
		if (x < width && y < height) {
			double wiSum = 0;
			double outputSum = 0;

			for (int i = 0; i < anchorPointsCount; i++) {
				const double wi = computeWiGpu(x, y, anchorPoints[i * 3], anchorPoints[i * 3 + 1], pParam);
				wiSum += wi;
				outputSum += wi * anchorPoints[i * 3 + 2];
			}
			outputSum /= wiSum;

			
			const auto outputSum8 = static_cast<uint8_t>(outputSum);
			surf2Dwrite(make_uchar1(outputSum8), surfObject, x, y);
		}
	}

	__global__ void gpuTextureColorKernel(const hipSurfaceObject_t input, hipSurfaceObject_t output, uchar4* colorData, const int width, const int height) {

		const int x = blockIdx.x * blockDim.x + threadIdx.x;
		const int y = blockIdx.y * blockDim.y + threadIdx.y;

		if (x < width && y < height) {

			uchar1 data;
			surf2Dread(&data, input, x, y);
			surf2Dwrite(colorData[data.x], output, x * 4, y);
		}
	}
}




GpuIdwTexture::GpuIdwTexture(const int _width, const int _height, const bool _useOpenGLInterop)
: GpuIdwBase(_width, _height, "GpuIdwTexture"), useOpenGLInterop(_useOpenGLInterop) {

	if (useOpenGLInterop) {
		initBasic();
		initWithInterop();
	} else {
		initBasic();
	}
	
}

GpuIdwTexture::~GpuIdwTexture() {

	if (greyscaleSurfObject)
		CHECK_ERROR(hipDestroySurfaceObject(greyscaleSurfObject));

	if (colorSurfObject)
		CHECK_ERROR(hipDestroySurfaceObject(colorSurfObject));
	
	if (cuArrayGreyscale)
		CHECK_ERROR(hipFreeArray(cuArrayGreyscale));

	if (cuArrayColor)
		CHECK_ERROR(hipFreeArray(cuArrayColor));
}

void GpuIdwTexture::refreshInnerGreyscaleGpu(const double pParam) {

	dim3 gridRes(width / 32, height / 32);
	dim3 blockRes(32, 32);

	//// Invoke kernel
	//dim3 dimBlock(16, 16);
	//dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x,
	//	(height + dimBlock.y - 1) / dimBlock.y);

	gpuTextureKernel << < gridRes, blockRes >> > (greyscaleSurfObject, anchorsGpu, anchorsGpuCurrentCount, pParam, width, height);
	CHECK_ERROR(hipGetLastError());
	CHECK_ERROR(hipDeviceSynchronize());
}

void GpuIdwTexture::refreshInnerGreyscaleDrawAnchorPoints(const std::vector<P2>& anchorPoints) {

	int power = 1;
	while (power < anchorsGpuCurrentCount)
		power *= 2;

	if (power >= 1024) {
		throw std::exception("power is bigger than 1024");
	}

	gpuDrawAnchorPointsKernel << < 1, power >> > (greyscaleSurfObject, anchorsGpu, anchorsGpuCurrentCount, width, height);
	CHECK_ERROR(hipGetLastError());
	CHECK_ERROR(hipDeviceSynchronize());
}


//// Invoke kernel
//dim3 dimBlock(16, 16);
//dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x,
//	(height + dimBlock.y - 1) / dimBlock.y);

void GpuIdwTexture::refreshInnerColorGpu() {

	dim3 gridRes(width / 32, height / 32);
	dim3 blockRes(32, 32);
	
	if (useOpenGLInterop) {
		CHECK_ERROR(hipGraphicsMapResources(1, &viewCudaResource));
		{
			hipArray_t viewCudaArray;
			CHECK_ERROR(hipGraphicsSubResourceGetMappedArray(&viewCudaArray, viewCudaResource, 0, 0));
			hipResourceDesc viewCudaArrayResourceDesc;
			viewCudaArrayResourceDesc.resType = hipResourceTypeArray;
			viewCudaArrayResourceDesc.res.array.array = viewCudaArray;

			hipSurfaceObject_t viewCudaSurfaceObject;
			CHECK_ERROR(hipCreateSurfaceObject(&viewCudaSurfaceObject, &viewCudaArrayResourceDesc));

			gpuTextureColorKernel << < gridRes, blockRes >> > (viewCudaSurfaceObject, colorSurfObject, colorMappingData, width, height);

			CHECK_ERROR(hipDestroySurfaceObject(viewCudaSurfaceObject));
			CHECK_ERROR(hipGetLastError());
			CHECK_ERROR(hipDeviceSynchronize());
		}
		CHECK_ERROR(hipGraphicsUnmapResources(1, &viewCudaResource));
		CHECK_ERROR(hipStreamSynchronize(nullptr));
		
	} else {
		gpuTextureColorKernel << < gridRes, blockRes >> > (greyscaleSurfObject, colorSurfObject, colorMappingData, width, height);
		CHECK_ERROR(hipGetLastError());
		CHECK_ERROR(hipDeviceSynchronize());
	}

}

void GpuIdwTexture::downloadGreyscaleBitmap() {
	CHECK_ERROR(hipMemcpyFromArray(bitmapGreyscaleCpu, cuArrayGreyscale, 0, 0, width * height * sizeof(uint8_t), hipMemcpyDeviceToHost));
}

void GpuIdwTexture::downloadColorBitmap() {
	CHECK_ERROR(hipMemcpyFromArray(bitmapColorCpu, cuArrayColor, 0, 0, width * height * sizeof(uchar4), hipMemcpyDeviceToHost));
}

void GpuIdwTexture::drawOpengl(DataManager& manager) {

	if (!useOpenGLInterop) {
		GpuIdwBase::drawOpengl(manager);
		return;
	}

	glMatrixMode(GL_PROJECTION);
	glPushMatrix();
	glLoadIdentity();
	glOrtho(0.0, glutGet(GLUT_WINDOW_WIDTH), 0.0, glutGet(GLUT_WINDOW_HEIGHT), -1.0, 1.0);
	glMatrixMode(GL_MODELVIEW);
	glPushMatrix();


	glLoadIdentity();
	glDisable(GL_LIGHTING);


	glColor3f(1, 1, 1);
	glEnable(GL_TEXTURE_2D);
	glBindTexture(GL_TEXTURE_2D, viewGLTexture);


	// Draw a textured quad
	glBegin(GL_QUADS);
	glTexCoord2f(0, 0); glVertex3f(0, 0, 0);
	glTexCoord2f(0, 1); glVertex3f(0, 100, 0);
	glTexCoord2f(1, 1); glVertex3f(100, 100, 0);
	glTexCoord2f(1, 0); glVertex3f(100, 0, 0);
	glEnd();


	glDisable(GL_TEXTURE_2D);
	glPopMatrix();


	glMatrixMode(GL_PROJECTION);
	glPopMatrix();

	glMatrixMode(GL_MODELVIEW);
}

void GpuIdwTexture::initBasic() {
	//greyscale
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);
	CHECK_ERROR(hipMallocArray(&cuArrayGreyscale, &channelDesc, width, height, hipArraySurfaceLoadStore));

	// Specify surface
	struct hipResourceDesc resDesc {};
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = cuArrayGreyscale;
	CHECK_ERROR(hipCreateSurfaceObject(&greyscaleSurfObject, &resDesc));

	//color
	channelDesc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
	CHECK_ERROR(hipMallocArray(&cuArrayColor, &channelDesc, width, height, hipArraySurfaceLoadStore));

	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = cuArrayColor;
	CHECK_ERROR(hipCreateSurfaceObject(&colorSurfObject, &resDesc));
}

void GpuIdwTexture::initWithInterop() {

	glEnable(GL_TEXTURE_2D);
	glGenTextures(1, &viewGLTexture);

	glBindTexture(GL_TEXTURE_2D, viewGLTexture);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, 768, 768, 0, GL_RGBA, GL_UNSIGNED_BYTE, nullptr);
	glBindTexture(GL_TEXTURE_2D, 0);

	CHECK_ERROR(hipGraphicsGLRegisterImage(&viewCudaResource, viewGLTexture, GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore));

}


