#include "GpuIdwBase.cuh"

static void handleCudaError(const hipError_t error, const char* file, const int line) {
	if (error == hipSuccess) return;

	fmt::print("{} in {} at line {}\n", hipGetErrorString(error), file, line);
	exit(EXIT_FAILURE);
}

#define CHECK_ERROR( error ) ( handleCudaError( error, __FILE__, __LINE__ ) )


GpuIdwBase::GpuIdwBase(const int _width, const int _height, const std::string& _methodName) : CpuIdwBase(_width, _height, _methodName) {

	CHECK_ERROR(hipMalloc(reinterpret_cast<void**>(&anchorsGpu), anchorsGpuBytes));
}

GpuIdwBase::~GpuIdwBase() {

	if (anchorsGpu) 
		CHECK_ERROR(hipFree(anchorsGpu));
}


void GpuIdwBase::refreshInner(const std::vector<P2>& anchorPoints, const double pParam) {
	lastVersionOnCpu = false;
	copyAnchorsToGpu(anchorPoints);
	refreshInnerGpu(pParam);
}

void GpuIdwBase::copyAnchorsToGpu(const std::vector<P2>& anchorPoints) {

	if (anchorPoints.size() > anchorsGpuMaxCount) {
		//free memory
		if (anchorsGpu) CHECK_ERROR(hipFree(anchorsGpu));

		anchorsGpuMaxCount = anchorsGpuMaxCount * 2;
		anchorsGpuBytes = anchorsGpuMaxCount * 3 * sizeof(int);

		//create bigger memory
		CHECK_ERROR(hipMalloc(reinterpret_cast<void**>(&anchorsGpu), anchorsGpuBytes));
	}

	anchorsGpuCurrentCount = anchorPoints.size();

	//i should be able to just read vector's data as ints ...

	const auto* rawPointer = reinterpret_cast<const int*>(anchorPoints.data());

	const auto err = hipMemcpy(anchorsGpu, rawPointer, anchorsGpuBytes, hipMemcpyHostToDevice);
	CHECK_ERROR(err);
}
