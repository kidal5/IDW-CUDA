#include "GpuIdwBase.cuh"

static void handleCudaError(const hipError_t error, const char* file, const int line) {
	if (error == hipSuccess) return;

	fmt::print("{} in {} at line {}\n", hipGetErrorString(error), file, line);
	exit(EXIT_FAILURE);
}

#define CHECK_ERROR( error ) ( handleCudaError( error, __FILE__, __LINE__ ) )


GpuIdwBase::GpuIdwBase(const int _width, const int _height, const std::string& _methodName) : IdwBase(_width, _height, _methodName) {

	imgBytesCount = width * height * 3 * sizeof(uint8_t);

	auto err = hipMalloc(reinterpret_cast<void**>(&bitmapGpu), imgBytesCount);
	CHECK_ERROR(err);

	err = hipMalloc(reinterpret_cast<void**>(&anchorsGpu), anchorsGpuBytes);
	CHECK_ERROR(err);
}

GpuIdwBase::~GpuIdwBase() {

	if (bitmapGpu)
		CHECK_ERROR(hipFree(bitmapGpu));

	if (anchorsGpu)
		CHECK_ERROR(hipFree(anchorsGpu));

}

uint8_t* GpuIdwBase::getBitmapCpu() {

	if (!lastVersionOnCpu) {
		auto err = hipMemcpy(bitmapCpu.get(), bitmapGpu, imgBytesCount, hipMemcpyDeviceToHost);

		lastVersionOnCpu = true;
	}

	return bitmapCpu.get();
}

void GpuIdwBase::refreshInner(const std::vector<P2>& anchorPoints, const double pParam) {
	lastVersionOnCpu = false;
	copyAnchorsToGpu(anchorPoints);
	refreshInnerGpu(pParam);
}

void GpuIdwBase::copyAnchorsToGpu(const std::vector<P2>& anchorPoints) {

	if (anchorPoints.size() > anchorsGpuMaxCount) {
		//free memory
		if (anchorsGpu) CHECK_ERROR(hipFree(anchorsGpu));

		anchorsGpuMaxCount = anchorsGpuMaxCount * 2;
		anchorsGpuBytes = anchorsGpuMaxCount * 3 * sizeof(int);

		//create bigger memory
		CHECK_ERROR(hipMalloc(reinterpret_cast<void**>(&anchorsGpu), anchorsGpuBytes));
	}

	anchorsGpuCurrentCount = anchorPoints.size();

	//i should be able to just read vector's data as ints ...

	const auto* rawPointer = reinterpret_cast<const int*>(anchorPoints.data());

	const auto err = hipMemcpy(anchorsGpu, rawPointer, anchorsGpuBytes, hipMemcpyHostToDevice);
	CHECK_ERROR(err);
}

