#include "hip/hip_runtime.h"
﻿#include "GpuBasic.cuh"

#include "hip/hip_runtime.h"
#include ""

namespace 
{
	__device__ double computeWiGpu(const int ax, const int ay, const int bx, const int by, double pParam) {
		const double dist = sqrt((ax - bx) * (ax - bx) + (ay - by) * (ay - by));
		return 1 / pow(dist, pParam);
	}
	
    __global__ void firstKernel(uint8_t* bitmap, const int * anchorPoints, const int anchorPointsCount, const double pParam, const int width, const int height) {

        const int xStart = blockIdx.x * blockDim.x + threadIdx.x;
		const int yStart = blockIdx.y * blockDim.y + threadIdx.y;

    	//fill its own chunk
		for (int h = yStart; h < yStart + blockDim.y && h < height; ++h) {
			for (int w = xStart; w < xStart + blockDim.x && w < width; ++w) {
				double wiSum = 0;
				double outputSum = 0;

				for (int i = 0; i < anchorPointsCount; i++){
					const double wi = computeWiGpu(w, h, anchorPoints[i * 3], anchorPoints[i * 3 + 1], pParam);
					wiSum += wi;
					outputSum += wi * anchorPoints[i * 3 + 2];
				}
				outputSum /= wiSum;

				bitmap[3 * (h * width + w) + 0] = static_cast<uint8_t>(outputSum);
				bitmap[3 * (h * width + w) + 1] = static_cast<uint8_t>(outputSum);
				bitmap[3 * (h * width + w) + 2] = static_cast<uint8_t>(outputSum);
			}
		}
    }
}


void GpuBasic::refreshInnerGpu(const double pParam) {
	dim3 blocks(768 / 16, 768 / 16);
	dim3 threads(16, 16);

	firstKernel < < < blocks, threads > > > (bitmapGpu, anchorsGpu, anchorsGpuCurrentCount, pParam, width, height);
	CHECK_ERROR(hipGetLastError());
}
